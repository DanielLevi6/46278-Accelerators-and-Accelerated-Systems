#include "hip/hip_runtime.h"
#include "ex2.h"
#include <cuda/atomic>

#define NO_ID (-1)

typedef struct{
    hipStream_t stream;

    bool available;

    uchar *target;
    uchar *reference;
    uchar *result;
    int job_id;
} stream_struct;

/***************************** Image processing funcs *****************************/
__device__ void prefixSum(int arr[], int len, int tid, int threads)
{
    //TODO
    int increment;
    
    for(int stride = 1; stride < threads; stride *= 2)
    {
        if(tid >= stride && tid < len)
        {
            increment = arr[tid - stride];
        }

        __syncthreads();

        if(tid >= stride && tid < len)
        {
            arr[tid] += increment;
        }

        __syncthreads();
    }
}

__device__ void argmin(int arr[], int len, int tid, int threads)
{
    assert(threads == len / 2);
    int halfLen = len / 2;
    bool firstIteration = true;
    int prevHalfLength = 0;
    while (halfLen > 0)
    {
        if (tid < halfLen)
        {
            if (arr[tid] == arr[tid + halfLen])
            { // a corner case
                int lhsIdx = tid;
                int rhdIdx = tid + halfLen;
                int lhsOriginalIdx = firstIteration ? lhsIdx : arr[prevHalfLength + lhsIdx];
                int rhsOriginalIdx = firstIteration ? rhdIdx : arr[prevHalfLength + rhdIdx];
                arr[tid + halfLen] = lhsOriginalIdx < rhsOriginalIdx ? lhsOriginalIdx : rhsOriginalIdx;
            }
            else
            { // the common case
                bool isLhsSmaller = (arr[tid] < arr[tid + halfLen]);
                int idxOfSmaller = isLhsSmaller * tid + (!isLhsSmaller) * (tid + halfLen);
                int smallerValue = arr[idxOfSmaller];
                int origIdxOfSmaller = firstIteration * idxOfSmaller + (!firstIteration) * arr[prevHalfLength + idxOfSmaller];
                arr[tid] = smallerValue;
                arr[tid + halfLen] = origIdxOfSmaller;
            }
        }
        __syncthreads();
        firstIteration = false;
        prevHalfLength = halfLen;
        halfLen /= 2;
    }
}

__device__ void colorHist(uchar img[][CHANNELS], int histograms[][LEVELS])
{
    //TODO
    int tid = threadIdx.x;
    int threads = blockDim.x;
    
    // Init the histograms
    if(tid < LEVELS)
    {
        histograms[0][tid] = 0; // Red channel
        histograms[1][tid] = 0; // Green channel
        histograms[2][tid] = 0; // Blue channel
    }

    __syncthreads();

    for(int i = tid; i < SIZE * SIZE; i += threads)
    {
        // Red channel
        atomicAdd_block(histograms[0] + img[i][0], 1);

        // Green channel
        atomicAdd_block(histograms[1] + img[i][1], 1);

        // Blue channel
        atomicAdd_block(histograms[2] + img[i][2], 1);
    }

    __syncthreads();
}

__device__ void performMapping(uchar maps[][LEVELS], uchar targetImg[][CHANNELS], uchar resultImg[][CHANNELS])
{
    //TODO
    int tid = threadIdx.x;
    int threads = blockDim.x;
    
    for(int i = tid; i < SIZE * SIZE; i += threads)
    {
        resultImg[i][0] = maps[0][targetImg[i][0]]; // Red channel
        resultImg[i][1] = maps[1][targetImg[i][1]]; // Green channel
        resultImg[i][2] = maps[2][targetImg[i][2]]; // Blue channel
    }

    __syncthreads();
}

__device__ void calculateMap(uchar maps[LEVELS], int targetHist[LEVELS], int refrenceHist[LEVELS])
{
    __shared__ int diff[LEVELS];

    int tid = threadIdx.x;
    int threads = blockDim.x;

    for(int i_tar = 0; i_tar < LEVELS; i_tar++){
        for(int i_ref = tid; i_ref < LEVELS; i_ref += threads){
            diff[i_ref] = abs(refrenceHist[i_ref] - targetHist[i_tar]);
        }

        __syncthreads();

        argmin(diff, LEVELS, tid, LEVELS/2);
        
        __syncthreads();

        if(tid == 0) // For preventing bank conflicts
        {
            maps[i_tar] = diff[1];
        }
        
        __syncthreads();
    }

}
/************************** Image processing funcs - end **************************/

__device__
void process_image(uchar *target, uchar *reference, uchar *result) {
    // TODO complete according to hw1
    __shared__ int targetHist[CHANNELS][LEVELS];
    __shared__ int refrenceHist[CHANNELS][LEVELS];
    __shared__ uchar maps[CHANNELS][LEVELS];

    int tid = threadIdx.x;
    int threads = blockDim.x;

    // Step 1 - For each image (target and reference), create a histogram
    colorHist(reinterpret_cast<uchar(*) [CHANNELS]>(target), targetHist);
    colorHist(reinterpret_cast<uchar(*) [CHANNELS]>(reference), refrenceHist);

    __syncthreads();
    
    // Step 2 - Calculate the prefix sum of the histogram
    for (int i = 0; i < CHANNELS; i++){
        prefixSum(targetHist[i], LEVELS, tid, threads);
        prefixSum(refrenceHist[i], LEVELS, tid, threads);
    }

    __syncthreads();

    // Step 3 - Calculate a map 𝑚 from old to new gray levels
    for (int i = 0; i < CHANNELS; i++){
        calculateMap(maps[i], targetHist[i], refrenceHist[i]);
    }

    __syncthreads();

    // Step 4 - Perform the mapping process
    performMapping(maps, reinterpret_cast<uchar(*)[CHANNELS]>(target), reinterpret_cast<uchar(*)[CHANNELS]>(result));
    
    __syncthreads();
}

__global__
void process_image_kernel(uchar *target, uchar *reference, uchar *result){
    process_image(target, reference, result);
}

class streams_server : public image_processing_server
{
private:
    // TODO define stream server context (memory buffers, streams, etc...)
    stream_struct stream_contexts[STREAM_COUNT];

public:
    streams_server()
    {
        // TODO initialize context (memory buffers, streams, etc...)
        for(int i=0; i<STREAM_COUNT; i++)
        {
            // Stream init
            CUDA_CHECK(hipStreamCreate(&stream_contexts[i].stream));

            // Buffer init
            CUDA_CHECK(hipMalloc(&stream_contexts[i].target, SIZE * SIZE * CHANNELS * sizeof(uchar)));
            CUDA_CHECK(hipMalloc(&stream_contexts[i].reference, SIZE * SIZE * CHANNELS * sizeof(uchar)));
            CUDA_CHECK(hipMalloc(&stream_contexts[i].result, SIZE * SIZE * CHANNELS * sizeof(uchar)));
            stream_contexts[i].job_id = NO_ID;

            // Availability init
            stream_contexts[i].available = true;
        }
    }

    ~streams_server() override
    {
        // TODO free resources allocated in constructor
        for(int i=0; i<STREAM_COUNT; i++)
        {
            // Stream free
            CUDA_CHECK(hipStreamDestroy(stream_contexts[i].stream));

            // Buffer free
            CUDA_CHECK(hipFree(stream_contexts[i].target));
            CUDA_CHECK(hipFree(stream_contexts[i].reference));
            CUDA_CHECK(hipFree(stream_contexts[i].result));
        }
    }

    bool enqueue(int job_id, uchar *target, uchar *reference, uchar *result) override
    {
        // TODO place memory transfers and kernel invocation in streams if possible.
        for(int i=0; i<STREAM_COUNT; i++)
        {
            if(stream_contexts[i].available)
            {
                stream_contexts[i].job_id = job_id;
                stream_contexts[i].available = false;
                CUDA_CHECK(hipMemcpyAsync(stream_contexts[i].target, target, SIZE * SIZE * CHANNELS * sizeof(uchar), hipMemcpyHostToDevice, stream_contexts[i].stream));
                CUDA_CHECK(hipMemcpyAsync(stream_contexts[i].reference, reference, SIZE * SIZE * CHANNELS * sizeof(uchar), hipMemcpyHostToDevice, stream_contexts[i].stream));
                process_image_kernel<<<1, 1024, 0, stream_contexts[i].stream>>>(stream_contexts[i].target, stream_contexts[i].reference, stream_contexts[i].result);
                CUDA_CHECK(hipMemcpyAsync(result, stream_contexts[i].result, SIZE * SIZE * CHANNELS * sizeof(uchar), hipMemcpyDeviceToHost, stream_contexts[i].stream));
                return true;
            }
        }
        return false;
    }

    bool dequeue(int *job_id) override
    {
        // TODO query (don't block) streams for any completed requests.
        for(int i=0; i<STREAM_COUNT; i++)
        {
            if(stream_contexts[i].job_id != NO_ID)
            {
                hipError_t status = hipStreamQuery(stream_contexts[i].stream); // TODO query diffrent stream each iteration
                switch (status) {
                case hipSuccess:
                    // TODO return the img_id of the request that was completed.
                    *job_id = stream_contexts[i].job_id;
                    stream_contexts[i].available = true;
                    stream_contexts[i].job_id = NO_ID;
                    return true;
                case hipErrorNotReady:
                    continue;
                default:
                    CUDA_CHECK(status);
                    return false;
                }
            }
        }
        return false;
    }
};

std::unique_ptr<image_processing_server> create_streams_server()
{
    return std::make_unique<streams_server>();
}

// TODO implement a SPSC queue
class SpscQueue
{

};

// template <typename T, uint8_t size> class ring_buffer {
// private:
//     static const size_t N = 1 << size;
//     T _mailbox[N];
//     cuda::atomic<size_t> _head = 0, _tail = 0;
// public:
//     void push(const T &data) {
//         int tail = _tail.load(memory_order_relaxed);
//         while (tail - _head.load(memory_order_acquire) == N);
//         _mailbox[_tail % N] = data;
//         _tail.store(tail + 1, memory_order_release);
//     }
//     T pop() {
//         int head = _head.load(memory_order_relaxed);
//         while (_tail.load(memory_order_acquire) == _head);
        
//         T item = _mailbox[_head % N];
//         _head.store(head + 1, memory_order_release);
//         return item;
//     }
// };

// TODO implement the persistent kernel
// TODO implement a function for calculating the threadblocks count

class queue_server : public image_processing_server
{
private:
    // TODO define queue server context (memory buffers, etc...)
    SpscQueue cpu_to_gpu_queue;
    SpscQueue gpu_to_cpu_queue;
public:
    queue_server(int threads)
    {
        // TODO initialize host state
        // TODO launch GPU persistent kernel with given number of threads, and calculated number of threadblocks
    }

    ~queue_server() override
    {
        // TODO free resources allocated in constructor
    }

    bool enqueue(int job_id, uchar *target, uchar *reference, uchar *result) override
    {
        // TODO push new task into queue if possible
        return false;
    }

    bool dequeue(int *job_id) override
    {
        // TODO query (don't block) the producer-consumer queue for any responses.
        return false;

        // TODO return the job_id of the request that was completed.
        //*job_id = ... 
        return true;
    }
};

std::unique_ptr<image_processing_server> create_queues_server(int threads)
{
    return std::make_unique<queue_server>(threads);
}
